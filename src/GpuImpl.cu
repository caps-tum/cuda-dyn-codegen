#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <array>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_new.h>
#include <thrust/device_delete.h>
#include <thrust/device_vector.h>

#include <boost/program_options.hpp>

#include "Matrix.h"
#include "GpuTimer.h"
#include "Stencil.h"
#include "Logger.h"
#include "Test.h"

__global__ void fivePoint1(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t width, size_t height) {
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y;

	size_t index = y * width + x;

	size_t left = index - 1;
	size_t right = index + 1;
	size_t top = index - width;
	size_t bottom = index + width;

	if (x >= width || y >= height) {
		return;
	}

	if (x < 1 || y < 1 || x == width - 1 || y == height - 1) {
		output[index] = input[index];
	}
	else {
		output[index] = 1.0f / 5 * (input[top] + input[left] + input[index] + input[right] + input[bottom]);
	}
}

__global__ void fivePoint2(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t width, size_t height) {
	size_t x = blockIdx.x * blockDim.x + threadIdx.x + 1;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y + 1;

	size_t index = y * width + x;

	size_t left = index - 1;
	size_t right = index + 1;
	size_t top = index - width;
	size_t bottom = index + width;

	if (x >= width - 1 || y >= height - 1) {
		return;
	}

	output[index] = 1.0f / 5 * (input[top] + input[left] + input[index] + input[right] + input[bottom]);
}

__global__ void fivePoint3(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t global_width, size_t global_height) {
	size_t global_x = blockIdx.x * blockDim.x + threadIdx.x + 1;
	size_t global_y = blockIdx.y * blockDim.y + threadIdx.y + 1;

	size_t global_index = global_y * global_width + global_x;

	size_t global_left = global_index - 1;
	size_t global_right = global_index + 1;
	size_t global_top = global_index - global_width;
	size_t global_bottom = global_index + global_width;

	extern __shared__ float buffer[];

	size_t buffer_x = threadIdx.x + 1;
	size_t buffer_y = threadIdx.y + 1;
	
	size_t buffer_width = 18;
	size_t buffer_height = 18;

	int buffer_index = buffer_y * buffer_width + buffer_x;

	size_t buffer_left = buffer_index - 1;
	size_t buffer_right = buffer_index + 1;
	size_t buffer_top = buffer_index - buffer_width;
	size_t buffer_bottom = buffer_index + buffer_width;

	buffer[buffer_index] = input[global_index];

	if (threadIdx.x == 0) {
		buffer[buffer_left] = input[global_left];
	}

	if (threadIdx.x == buffer_width - 3) {
		buffer[buffer_right] = input[global_right];
	}

	if (threadIdx.y == 0) {
		buffer[buffer_top] = input[global_top];
	}

	if (threadIdx.y == buffer_height - 3) {
		buffer[buffer_bottom] = input[global_bottom];
	}
	__syncthreads();

	output[global_index] = 1.0f / 5 * (buffer[buffer_top] + buffer[buffer_left] + buffer[buffer_index] + buffer[buffer_right] + buffer[buffer_bottom]);
}


__global__ void fivePoint3_2(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t global_width, size_t global_height) {
	size_t global_x = blockIdx.x * blockDim.x + threadIdx.x + 1;
	size_t global_y = blockIdx.y * blockDim.y + threadIdx.y + 1;

	size_t global_index = global_y * global_width + global_x;

	size_t global_left = global_index - 1;
	size_t global_right = global_index + 1;
	size_t global_top = global_index - global_width;
	size_t global_bottom = global_index + global_width;

	__shared__ float buffer[18][18];

	size_t buffer_x = threadIdx.x + 1;
	size_t buffer_y = threadIdx.y + 1;

	size_t buffer_width = 18;
	size_t buffer_height = 18;

	int buffer_index = buffer_y * buffer_width + buffer_x;

	size_t buffer_left = buffer_index - 1;
	size_t buffer_right = buffer_index + 1;
	size_t buffer_top = buffer_index - buffer_width;
	size_t buffer_bottom = buffer_index + buffer_width;

	if (threadIdx.x == 0) {
	buffer[0][buffer_y] = input[global_left];
	}

	if (threadIdx.x == buffer_width - 3) {
	buffer[buffer_width - 1][buffer_y] = input[global_right];
	}

	if (threadIdx.y == 0) {
	buffer[buffer_x][0] = input[global_top];
	}

	if (threadIdx.y == buffer_height - 3) {
	buffer[buffer_x][buffer_height - 1] = input[global_bottom];
	}

	buffer[buffer_x][buffer_y] = input[global_index];

	__syncthreads();

	output[global_index] = 1.0f / 5 * (buffer[buffer_x][buffer_y - 1] + buffer[buffer_x - 1][buffer_y] + buffer[buffer_x][buffer_y] + buffer[buffer_x + 1][buffer_y] + buffer[buffer_x][buffer_y + 1]);
}


__global__ void ninePoint1(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t width, size_t height, thrust::device_ptr<int const> const weights) {
	size_t x = blockIdx.x * blockDim.x + threadIdx.x + 2;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y + 2;

	size_t index = y * width + x;

	if (x >= width - 2 || y >= height - 2) {
		return;
	}

	output[index] = 1.0f / 9 * (
		weights[0 * 5 + 2] * input[index - width - width] +
		weights[1 * 5 + 2] * input[index - width] +
		weights[2 * 5 + 0] * input[index - 1 - 1] +
		weights[2 * 5 + 1] * input[index - 1] +
		weights[2 * 5 + 2] * input[index] +
		weights[2 * 5 + 3] * input[index + 1] +
		weights[2 * 5 + 4] * input[index + 1 + 1] +
		weights[3 * 5 + 2] * input[index + width] +
		weights[4 * 5 + 2] * input[index + width + width]
	);
}



std::array<int, 5> sizes { 512 + 2, 1024 + 2, 2048 + 2, 4096 + 2, 8192 + 2 };
std::array<size_t, 5> results;

size_t iterationsPerSize = 60;

//std::array<int, 1> sizes { 32 + 2 };
//std::array<size_t, 1> results;

//size_t iterationsPerSize = 3;




void test1() {
	std::fill(std::begin(results), std::end(results), 0);

	// jeweils 2x
	for (auto k = 0; k < 2; ++k) {
		// für alle Größen
		for (auto s = 0; s < sizes.size(); ++s) {
			auto size = sizes[s];
			
			auto input = thrust::device_new<float>(size * size);
			auto output = thrust::device_new<float>(size * size);

			Matrix<float> data(size, size);

			thrust::copy_n(data.raw(), size * size, input);

			dim3 blockSize { 16, 16, 1 };
			dim3 gridSize { (size + blockSize.x - 1) / blockSize.x, (size + blockSize.y - 1) / blockSize.y, 1 };

			GpuTimer timer;

			timer.start();

			for (auto i = 0; i < iterationsPerSize; ++i) {
				fivePoint1<<<gridSize, blockSize>>>(input, output, size, size);
				thrust::swap(input, output);
			}

			timer.stop();

			results[s] += stencilsPerSecond(size, size, timer.getDuration()) / iterationsPerSize;

			thrust::copy_n(input, size * size, data.raw());

			if (size == 128) {
				std::ofstream file("data.txt");
				file << data;
				file.close();
			}
			
			thrust::device_delete(output, size * size);
			thrust::device_delete(input, size * size);			
		}
	}

	std::transform(std::begin(results), std::end(results), std::begin(results), [](size_t r) { return r / 2; });

	Logger csv;
	csv.log("GpuImpl1");
	csv.log("Size", "Stencils/Second");

	for (auto i = 0; i < sizes.size(); ++i) {
		csv.log(sizes[i], results[i]);
	}

	std::ofstream file("gpu-impl1.csv");
	csv.writeTo(file);
	file.close();
}

void test2() {
	std::fill(std::begin(results), std::end(results), 0);

	// jeweils 2x
	for (auto k = 0; k < 2; ++k) {
		// für alle Größen
		for (auto s = 0; s < sizes.size(); ++s) {
			auto size = sizes[s];

			auto input = thrust::device_new<float>(size * size);
			auto output = thrust::device_new<float>(size * size);

			Matrix<float> data(size, size);

			thrust::copy_n(data.raw(), size * size, input);
			thrust::copy_n(data.raw(), size * size, output);

			dim3 blockSize { 16, 16, 1 };
			dim3 gridSize { (size + blockSize.x - 1 - 2) / blockSize.x, (size + blockSize.y - 1 - 2) / blockSize.y, 1 };

			GpuTimer timer;

			timer.start();

			for (auto i = 0; i < iterationsPerSize; ++i) {
				fivePoint2<<<gridSize, blockSize>>>(input, output, size, size);
				thrust::swap(input, output);
			}

			timer.stop();

			results[s] += stencilsPerSecond(size, size, timer.getDuration()) / iterationsPerSize;

			thrust::copy_n(input, size * size, data.raw());

			/*if (size == 35 + 2) {
				std::ofstream file("data.txt");
				file << data;
				file.close();
			}*/

			thrust::device_delete(output, size * size);
			thrust::device_delete(input, size * size);
		}
	}

	std::transform(std::begin(results), std::end(results), std::begin(results), [](size_t r) { return r / 2; });

	Logger csv;
	csv.log("GpuImpl2");
	csv.log("Size", "Stencils/Second");

	for (auto i = 0; i < sizes.size(); ++i) {
		csv.log(sizes[i], results[i]);
	}

	std::ofstream file("gpu-impl2.csv");
	csv.writeTo(file);
	file.close();
}

void test3() {
	std::fill(std::begin(results), std::end(results), 0);

	// jeweils 2x
	for (auto k = 0; k < 2; ++k) {
		// für alle Größen
		for (auto s = 0; s < sizes.size(); ++s) {
			auto size = sizes[s];

			auto input = thrust::device_new<float>(size * size);
			auto output = thrust::device_new<float>(size * size);

			Matrix<float> data(size, size);

			thrust::copy_n(data.raw(), size * size, input);
			thrust::copy_n(data.raw(), size * size, output);

			dim3 blockSize { 16, 16, 1 };
			dim3 gridSize { (size + blockSize.x - 1 - 2) / blockSize.x, (size + blockSize.y - 1 - 2) / blockSize.y, 1 };

			GpuTimer timer;

			timer.start();

			for (auto i = 0; i < iterationsPerSize; ++i) {
				//fivePoint3<<<gridSize, blockSize, sizeof(float) * 18 * 18>>>(input, output, size, size);
				fivePoint3_2 << <gridSize, blockSize>> >(input, output, size, size);
				thrust::swap(input, output);
			}

			timer.stop();

			results[s] += stencilsPerSecond(size, size, timer.getDuration()) / iterationsPerSize;

			thrust::copy_n(input, size * size, data.raw());

			if (size == 32 + 2) {
				std::ofstream file("data.txt");
				file << data;
				file.close();
			}

			thrust::device_delete(output, size * size);
			thrust::device_delete(input, size * size);
		}
	}

	std::transform(std::begin(results), std::end(results), std::begin(results), [](size_t r) { return r / 2; });

	Logger csv;
	csv.log("GpuImpl3");
	csv.log("Size", "Stencils/Second");

	for (auto i = 0; i < sizes.size(); ++i) {
		csv.log(sizes[i], results[i]);
	}

	std::ofstream file("gpu-impl3.csv");
	csv.writeTo(file);
	file.close();
}

void runCpu(boost::program_options::variables_map const& vm) {

}

void runGpu(boost::program_options::variables_map const& vm) {
	switch (vm["kernel"].as<int>()) {
	case 6: {
		auto result = 0;
		auto size = 32 + 4;
		auto input = thrust::device_new<float>(size * size);
		auto output = thrust::device_new<float>(size * size);

		Matrix<float> data(size, size);

		thrust::copy_n(data.raw(), size * size, input);
		thrust::copy_n(data.raw(), size * size, output);

		dim3 blockSize { 16, 16, 1 };
		dim3 gridSize { (size + blockSize.x - 1 - 4) / blockSize.x, (size + blockSize.y - 1 - 4) / blockSize.y, 1 };

		GpuTimer timer;

		timer.start();

		thrust::device_vector<int> weights = std::vector<int> {
			 0, 0, 1, 0, 0,
			 0, 0, 1, 0, 0,
			 1, 1, 1, 1, 1,
			 0, 0, 1, 0, 0,
			 0, 0, 1, 0, 0
		};	
		

		for (auto i = 0; i < iterationsPerSize; ++i) {
			//fivePoint3<<<gridSize, blockSize, sizeof(float) * 18 * 18>>>(input, output, size, size);
			ninePoint1 << <gridSize, blockSize >> >(input, output, size, size, weights.data());
			thrust::swap(input, output);
		}

		timer.stop();

		result += stencilsPerSecond(size, size, timer.getDuration()) / iterationsPerSize;

		thrust::copy_n(input, size * size, data.raw());

		std::cout << result;

		if (size == 32 + 4) {
			std::ofstream file("data.txt");
			file << data;
			file.close();
		}

		thrust::device_delete(output, size * size);
		thrust::device_delete(input, size * size);
	}
	default:
		break;
	}
}


void parseParameters(int argc, char* argv[]) {
	namespace po = boost::program_options;

	po::options_description general("General Options");

	std::string type;

	general.add_options()
		("type", po::value<std::string>(&type), "\"cpu\" or \"gpu\"")
		("width", po::value<size_t>()->default_value(1024), "width of the matrix")
		("height", po::value<size_t>()->default_value(1024), "height of the matrix")
		("numIterations", po::value<size_t>()->default_value(50), "number of iterations to calculate")
		("output", po::value<std::string>()->default_value("output.csv"), "name of the output file")
	;

	po::options_description gpu("GPU Options");

	gpu.add_options()
		("kernel", po::value<int>()->default_value(1), "version of the kernel to use")
	;

	po::options_description all("Usage");

	all.add(general).add(gpu);

	po::variables_map vm;
	po::store(po::parse_command_line(argc, argv, all), vm);
	po::notify(vm);

	if (type == "cpu") {
		runCpu(vm);
	}
	else if (type == "gpu") {
		runGpu(vm);
	}
	else {
		std::cout << "Error: --type must be set to either cpu or gpu.\n\n";
		std::cout << all << std::endl;
		//return 0;
	}
}


using NormalTest = Test<size_t, size_t>;


int main(int argc, char* argv[]) {
	//test2();
	//test1();
	//test3();


	//NormalTest t1(fivePoint1, dim3(258, 258), dim3(16, 16), 1);
	//t1.run(258, 258);

	parseParameters(argc, argv);

	std::cout << "so";
	std::cin.ignore();
}

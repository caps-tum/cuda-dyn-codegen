#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <array>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_new.h>
#include <thrust/device_delete.h>

#include <boost/program_options.hpp>

#include "Matrix.h"
#include "GpuTimer.h"
#include "Stencil.h"
#include "Logger.h"

namespace po = boost::program_options;

__global__ void fivePoint1(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t width, size_t height) {
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y;

	size_t index = y * width + x;

	size_t left = index - 1;
	size_t right = index + 1;
	size_t top = index - width;
	size_t bottom = index + width;

	if (x >= width || y >= height) {
		return;
	}

	if (x < 1 || y < 1 || x == width - 1 || y == height - 1) {
		output[index] = input[index];
	}
	else {
		output[index] = 1.0f / 5 * (input[top] + input[left] + input[index] + input[right] + input[bottom]);
	}
}

__global__ void fivePoint2(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t width, size_t height) {
	size_t x = blockIdx.x * blockDim.x + threadIdx.x + 1;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y + 1;
	//size_t width = gridDim.x * blockDim.x + 2;
	//size_t height = gridDim.y * blockDim.y + 2;
	
	size_t index = y * width + x;

	size_t left = index - 1;
	size_t right = index + 1;
	size_t top = index - width;
	size_t bottom = index + width;

	if (x >= width - 1 || y >= height - 1) {
		return;
	}

	//output[index] = 1.0f / 5 * (input[left] + input[index] + input[right]+ input[top] + input[bottom]);
	// Wenn man die Zeile oben stattdessen so umsortiert, dass die zu addierenden Elemente in der Reihenfolge
	// aufsteigender Speicheradressen im Term auftauchen, gibt es einen großen Geschwindigkeitsschub.
	// 1380000 -> 1630000.
	output[index] = 1.0f / 5 * (input[top] + input[left] + input[index] + input[right] + input[bottom]);
}

__global__ void fivePoint3(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t global_width, size_t global_height) {
	size_t global_x = blockIdx.x * blockDim.x + threadIdx.x + 1;
	size_t global_y = blockIdx.y * blockDim.y + threadIdx.y + 1;

	size_t global_index = global_y * global_width + global_x;

	size_t global_left = global_index - 1;
	size_t global_right = global_index + 1;
	size_t global_top = global_index - global_width;
	size_t global_bottom = global_index + global_width;

	extern __shared__ float buffer[];

	size_t buffer_x = threadIdx.x + 1;
	size_t buffer_y = threadIdx.y + 1;
	
	size_t buffer_width = 18;
	size_t buffer_height = 18;

	int buffer_index = buffer_y * buffer_width + buffer_x;

	size_t buffer_left = buffer_index - 1;
	size_t buffer_right = buffer_index + 1;
	size_t buffer_top = buffer_index - buffer_width;
	size_t buffer_bottom = buffer_index + buffer_width;

	buffer[buffer_index] = input[global_index];

	if (threadIdx.x == 0) {
		buffer[buffer_left] = input[global_left];
	}

	if (threadIdx.x == buffer_width - 3) {
		buffer[buffer_right] = input[global_right];
	}

	if (threadIdx.y == 0) {
		buffer[buffer_top] = input[global_top];
	}

	if (threadIdx.y == buffer_height - 3) {
		buffer[buffer_bottom] = input[global_bottom];
	}
	__syncthreads();

	// Wenn man hier umsortiert, bringt das komischerweise gar nichts.
	// Allerdings ist er schneller, wenn man stattdessen die Variante mit dem 2d-Array benutzt, vielleicht kann er
	// dann besser optimieren.
	output[global_index] = 1.0f / 5 * (buffer[buffer_left] + buffer[buffer_index] + buffer[buffer_right] + buffer[buffer_top] + buffer[buffer_bottom]);
}


__global__ void fivePoint3_2(thrust::device_ptr<float const> const input, thrust::device_ptr<float> const output, size_t global_width, size_t global_height) {
	size_t global_x = blockIdx.x * blockDim.x + threadIdx.x + 1;
	size_t global_y = blockIdx.y * blockDim.y + threadIdx.y + 1;

	size_t global_index = global_y * global_width + global_x;

	size_t global_left = global_index - 1;
	size_t global_right = global_index + 1;
	size_t global_top = global_index - global_width;
	size_t global_bottom = global_index + global_width;

	__shared__ float buffer[18][18];

	size_t buffer_x = threadIdx.x + 1;
	size_t buffer_y = threadIdx.y + 1;

	size_t buffer_width = 18;
	size_t buffer_height = 18;

	int buffer_index = buffer_y * buffer_width + buffer_x;

	size_t buffer_left = buffer_index - 1;
	size_t buffer_right = buffer_index + 1;
	size_t buffer_top = buffer_index - buffer_width;
	size_t buffer_bottom = buffer_index + buffer_width;

	if (threadIdx.x == 0) {
	buffer[0][buffer_y] = input[global_left];
	}

	if (threadIdx.x == buffer_width - 3) {
	buffer[buffer_width - 1][buffer_y] = input[global_right];
	}

	if (threadIdx.y == 0) {
	buffer[buffer_x][0] = input[global_top];
	}

	if (threadIdx.y == buffer_height - 3) {
	buffer[buffer_x][buffer_height - 1] = input[global_bottom];
	}

	buffer[buffer_x][buffer_y] = input[global_index];

	__syncthreads();

	// Wenn man hier umsortiert, bringt das komischerweise gar nichts.
	// Allerdings ist er schneller, wenn man stattdessen die Variante mit dem 2d-Array benutzt, vielleicht kann er
	// dann besser optimieren.
	output[global_index] = 1.0f / 5 * (buffer[buffer_x][buffer_y - 1] + buffer[buffer_x - 1][buffer_y] + buffer[buffer_x][buffer_y] + buffer[buffer_x + 1][buffer_y] + buffer[buffer_x][buffer_y + 1]);
}

std::array<int, 5> sizes { 512 + 2, 1024 + 2, 2048 + 2, 4096 + 2, 8192 + 2 };
std::array<size_t, 5> results;

size_t iterationsPerSize = 60;

//std::array<int, 1> sizes { 32 + 2 };
//std::array<size_t, 1> results;

//size_t iterationsPerSize = 3;


void test1() {
	std::fill(std::begin(results), std::end(results), 0);

	// jeweils 2x
	for (auto k = 0; k < 2; ++k) {
		// für alle Größen
		for (auto s = 0; s < sizes.size(); ++s) {
			auto size = sizes[s];
			
			auto input = thrust::device_new<float>(size * size);
			auto output = thrust::device_new<float>(size * size);

			Matrix<float> data(size, size);

			thrust::copy_n(data.raw(), size * size, input);

			dim3 blockSize { 16, 16, 1 };
			dim3 gridSize { (size + blockSize.x - 1) / blockSize.x, (size + blockSize.y - 1) / blockSize.y, 1 };

			GpuTimer timer;

			timer.start();

			for (auto i = 0; i < iterationsPerSize; ++i) {
				fivePoint1<<<gridSize, blockSize>>>(input, output, size, size);
				thrust::swap(input, output);
			}

			timer.stop();

			results[s] += stencilsPerSecond(size, size, timer.getDuration()) / iterationsPerSize;

			thrust::copy_n(input, size * size, data.raw());

			if (size == 128) {
				std::ofstream file("data.txt");
				file << data;
				file.close();
			}
			
			thrust::device_delete(output, size * size);
			thrust::device_delete(input, size * size);			
		}
	}

	std::transform(std::begin(results), std::end(results), std::begin(results), [](size_t r) { return r / 2; });

	Logger csv;
	csv.log("GpuImpl1");
	csv.log("Size", "Stencils/Second");

	for (auto i = 0; i < sizes.size(); ++i) {
		csv.log(sizes[i], results[i]);
	}

	std::ofstream file("gpu-impl1.csv");
	csv.writeTo(file);
	file.close();
}

void test2() {
	std::fill(std::begin(results), std::end(results), 0);

	// jeweils 2x
	for (auto k = 0; k < 2; ++k) {
		// für alle Größen
		for (auto s = 0; s < sizes.size(); ++s) {
			auto size = sizes[s];

			auto input = thrust::device_new<float>(size * size);
			auto output = thrust::device_new<float>(size * size);

			Matrix<float> data(size, size);

			thrust::copy_n(data.raw(), size * size, input);
			thrust::copy_n(data.raw(), size * size, output);

			dim3 blockSize { 16, 16, 1 };
			dim3 gridSize { (size + blockSize.x - 1 - 2) / blockSize.x, (size + blockSize.y - 1 - 2) / blockSize.y, 1 };

			GpuTimer timer;

			timer.start();

			for (auto i = 0; i < iterationsPerSize; ++i) {
				fivePoint2<<<gridSize, blockSize>>>(input, output, size, size);
				thrust::swap(input, output);
			}

			timer.stop();

			results[s] += stencilsPerSecond(size, size, timer.getDuration()) / iterationsPerSize;

			thrust::copy_n(input, size * size, data.raw());

			/*if (size == 35 + 2) {
				std::ofstream file("data.txt");
				file << data;
				file.close();
			}*/

			thrust::device_delete(output, size * size);
			thrust::device_delete(input, size * size);
		}
	}

	std::transform(std::begin(results), std::end(results), std::begin(results), [](size_t r) { return r / 2; });

	Logger csv;
	csv.log("GpuImpl2");
	csv.log("Size", "Stencils/Second");

	for (auto i = 0; i < sizes.size(); ++i) {
		csv.log(sizes[i], results[i]);
	}

	std::ofstream file("gpu-impl2.csv");
	csv.writeTo(file);
	file.close();
}

void test3() {
	std::fill(std::begin(results), std::end(results), 0);

	// jeweils 2x
	for (auto k = 0; k < 2; ++k) {
		// für alle Größen
		for (auto s = 0; s < sizes.size(); ++s) {
			auto size = sizes[s];

			auto input = thrust::device_new<float>(size * size);
			auto output = thrust::device_new<float>(size * size);

			Matrix<float> data(size, size);

			thrust::copy_n(data.raw(), size * size, input);
			thrust::copy_n(data.raw(), size * size, output);

			dim3 blockSize { 16, 16, 1 };
			dim3 gridSize { (size + blockSize.x - 1 - 2) / blockSize.x, (size + blockSize.y - 1 - 2) / blockSize.y, 1 };

			GpuTimer timer;

			timer.start();

			for (auto i = 0; i < iterationsPerSize; ++i) {
				//fivePoint3<<<gridSize, blockSize, sizeof(float) * 18 * 18>>>(input, output, size, size);
				fivePoint3_2 << <gridSize, blockSize>> >(input, output, size, size);
				thrust::swap(input, output);
			}

			timer.stop();

			results[s] += stencilsPerSecond(size, size, timer.getDuration()) / iterationsPerSize;

			thrust::copy_n(input, size * size, data.raw());

			if (size == 32 + 2) {
				std::ofstream file("data.txt");
				file << data;
				file.close();
			}

			thrust::device_delete(output, size * size);
			thrust::device_delete(input, size * size);
		}
	}

	std::transform(std::begin(results), std::end(results), std::begin(results), [](size_t r) { return r / 2; });

	Logger csv;
	csv.log("GpuImpl3");
	csv.log("Size", "Stencils/Second");

	for (auto i = 0; i < sizes.size(); ++i) {
		csv.log(sizes[i], results[i]);
	}

	std::ofstream file("gpu-impl3.csv");
	csv.writeTo(file);
	file.close();
}

int main(int argc, char* argv[]) {
	//test2();
	//test1();
	//test3();

	po::options_description desc("Allowed options");

	desc.add_options()
		("help", "produce help message")
		("kernel", po::value<int>(), "set the kernel version to use")
		;

	po::variables_map vm;
	po::store(po::parse_command_line(argc, argv, desc), vm);
	po::notify(vm);


	std::cin.ignore();
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

#include "Cuda.h"
#include "GpuTimer.h"

char const* source =
"extern \"C\" __global__ void inc(float const* input, float* output) {"
"       output[threadIdx.y * blockDim.y + threadIdx.x] = input[threadIdx.y * blockDim.y + threadIdx.x] + 1;"
"}"
;

void call(hiprtcResult result) {
	if (result != HIPRTC_SUCCESS) {
		std::cerr << "\nerror: " << result << " failed with error "
			<< hiprtcGetErrorString(result) << '\n';
		exit(1);
	}
}

void call(hipError_t result) {
	if (result != hipSuccess) {
		const char *msg;
		hipDrvGetErrorName(result, &msg);
		std::cerr << "\nerror: " << result << " failed with error "
			<< msg << '\n';
		exit(1);
	}
}


Cuda cuda;

struct Kernel {
        hipFunction_t kernel;

        void launch(dim3 gridSize, dim3 blockSize, hipDeviceptr_t in, hipDeviceptr_t out) {
                void* args[] = { &in, &out };

                call(hipModuleLaunchKernel(kernel, gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z, 0, nullptr, args, 0));
        }
};


int main() {
	GpuTimer timer;

	timer.start();

        hiprtcProgram prog;
        hiprtcCreateProgram(&prog, source, "source.cu", 0, nullptr, nullptr);

        hiprtcCompileProgram(prog, 0, nullptr);

        //size_t logSize;
        //hiprtcGetProgramLogSize(prog, &logSize);

        //char log[logSize];

        //hiprtcGetProgramLog(prog, log);

        //std::cout << "\n\nLOG:\n" << "====\n" << log << std::endl;

        size_t ptxSize;
        hiprtcGetCodeSize(prog, &ptxSize);

        char ptx[ptxSize];

        call(hiprtcGetCode(prog, ptx));

        hiprtcDestroyProgram(&prog);

        hipModule_t module;
        call(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));

        hipFunction_t kernel;
        call(hipModuleGetFunction(&kernel, module, "inc"));

        hipDeviceptr_t data;
        hipMalloc(&data, 5 * 5 * sizeof(float));

        Kernel k = { kernel };
        k.launch(dim3(1, 1, 1), dim3(5, 5, 1), data, data);

        call(hipCtxSynchronize());

        float* dataH = new float[5 * 5];

        call(hipMemcpyDtoH(dataH, data, 5 * 5 * sizeof(float)));

	timer.stop();

        for (auto i = 0; i < 5 * 5; i++) {
                printf("%f; ", dataH[i]);
        }

	std::cout << "\n\n" << timer.getDuration().count() << " us\n";
}

